#include "hip/hip_runtime.h"
// Translated into C++ from Java, based on the code available at: http://physics.princeton.edu/~fpretori/Nbody/
#define _USE_MATH_DEFINES
#include <cmath>
#include "hip/hip_runtime.h"
#include ""
#include "body.h"
#include <vector>
#include <allegro5/allegro.h>
#include <allegro5/allegro_primitives.h>
#include <allegro5/allegro_color.h>
#include <allegro5/allegro_font.h>
#include <allegro5/allegro_ttf.h>
#include <iostream>
#include <chrono>
#include <string>
using namespace std;
const double G = 6.673e-11;   // gravitational constant
int N = 1024;
vector<Body*> bodies; //Body bodies[1000];

int screen_size_x = 1024;
int screen_size_y = 768;
#define BLOCK_SIZE 512

//struct force
//{
//	double fx;
//	double fy;
//};

struct Body_struct
{
	double vx;
	double vy;
	double rx;
	double ry;
	double mass;
};

__global__ void add_force_block_doubles(double *vx, double *vy, const double *rx, const double *ry,
	const double *mass, double *r_rx, double *r_ry, const double G, int N, double dt)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < N)
	{
		// Reset forces
		double fx, fy = 0.0;

		for (int tile = 0; tile < gridDim.x; tile++)
		{
			__shared__ double s_rx[BLOCK_SIZE];
			__shared__ double s_ry[BLOCK_SIZE];
			double t_rx = rx[tile * blockDim.x + threadIdx.x];
			double t_ry = ry[tile * blockDim.x + threadIdx.x];
			s_rx[threadIdx.x] = t_rx;
			s_ry[threadIdx.x] = t_ry;
			__syncthreads();
			for (int j = 0; j < BLOCK_SIZE; j++)
			{
				if (idx != j)
				{
					double EPS = 3E4;      // softening parameter (just to avoid infinities)
					double dx = s_rx[j] - rx[idx];
					double dy = s_ry[j] - ry[idx];
					double dist = sqrt(dx*dx + dy*dy);
					double F = (G * mass[idx] * mass[j]) / (dist*dist + EPS*EPS);

					fx += F * dx / dist;
					fy += F * dy / dist;
				}
				__syncthreads();
			}
			//__syncthreads();
		}

		// Calculate velocity and integrate
		vx[idx] += dt * fx / mass[idx];
		vy[idx] += dt * fy / mass[idx];
		r_rx[idx] += dt * vx[idx];
		r_ry[idx] += dt * vy[idx];
	}
}

__global__ void add_force(const Body *bodies, double *vx, double *vy, const double G, int N, double dt)
{
	// Get block index
	unsigned int block_idx = blockIdx.x;
	// Get thread index
	unsigned int thread_idx = threadIdx.x;
	// Get the number of threads per block
	unsigned int block_dim = blockDim.x;
	// Get the thread's unique ID = (block_idx * block_dim) + thread_idx;
	unsigned int idx = (block_idx * block_dim) + thread_idx;
	if (idx < N)
	{
		// Reset forces
		double fx, fy = 0.0;
		//bodies[idx].fx = 0.0;
		//bodies[idx].fy = 0.0;

		for (int tile = 0; tile < gridDim.x; tile++)
		{
			__shared__ double s_rx[BLOCK_SIZE];
			__shared__ double s_ry[BLOCK_SIZE];
			double t_rx = bodies[tile * blockDim.x + threadIdx.x].rx;
			double t_ry = bodies[tile * blockDim.x + threadIdx.x].ry;
			s_rx[threadIdx.x] = t_rx;
			s_ry[threadIdx.x] = t_ry;
			__syncthreads();
			for (int j = 0; j < BLOCK_SIZE; j++)
			{
				if (idx != j)
				{
					double EPS = 3E4;      // softening parameter (just to avoid infinities)
					double dx = s_rx[j] - bodies[idx].rx;
					double dy = s_ry[j] - bodies[idx].ry;
					double dist = sqrt(dx*dx + dy*dy);
					double F = (G * bodies[idx].mass * bodies[j].mass) / (dist*dist + EPS*EPS);

					// is this right? probably
					//f[idx].fx += F * dx / dist;
					//f[idx].fy += F * dy / dist;

					fx += F * dx / dist;
					fy += F * dy / dist;
				}
				
			}
			__syncthreads();
		}
		
		// Calculate velocity
		vx[idx] += dt * fx / bodies[idx].mass;
		vy[idx] += dt * fy / bodies[idx].mass;
	}
	//for (int j = 0; j < N; j++) 
	//{
	//	if (idx != j)
	//	{
	//		double EPS = 3E4;      // softening parameter (just to avoid infinities)
	//		double dx = bodies[j].rx - bodies[idx].rx;
	//		double dy = bodies[j].ry - bodies[idx].ry;
	//		double dist = sqrt(dx*dx + dy*dy);
	//		double F = (G * bodies[idx].mass * bodies[j].mass) / (dist*dist + EPS*EPS);

	//		//fx[idx] = bodies[idx].fx;
	//		//fy[idx] = bodies[j].fx;
	//		f[idx].fx += F * dx / dist;
	//		f[idx].fy += F * dy / dist;
	//		//bodies_return[idx].fx += F * dx / dist;
	//		//bodies_return[idx].fy += F * dy / dist;
	//	}
	//}

	// Update velocity
	//bodies_return[idx].vx += dt * bodies_return[idx].fx / bodies_return[idx].mass;
	//bodies_return[idx].vy += dt * bodies_return[idx].fy / bodies_return[idx].mass;
	//bodies_return[idx].rx += dt * bodies_return[idx].vx;
	//bodies_return[idx].ry += dt * bodies_return[idx].vy;
	//c->fx += F * dx / dist;
	//c->fy += F * dy / dist;
}

__global__ void add_force_simple(const Body *bodies, Body *returned_bodies, const double G, int N, double dt)//(Body *bodies, double *vx, double *vy, const double G, int N, double dt)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < N)
	{
		double fx, fy = 0;
		returned_bodies[idx].fx = 0;
		returned_bodies[idx].fy = 0;
		for (int j = 0; j < N; j++)
		{
			if (idx != j)
			{
				double EPS = 3E4;      // softening parameter (just to avoid infinities)
				double dx = bodies[j].rx - bodies[idx].rx;
				double dy = bodies[j].ry - bodies[idx].ry;
				double dist = sqrt(dx*dx + dy*dy);
				double F = (G * bodies[idx].mass * bodies[j].mass) / (dist*dist + EPS*EPS);

				fx += F * dx / dist;
				fy += F * dy / dist;
				//returned_bodies[idx].fx += fx;
				//returned_bodies[idx].fy += fy;
				//fx[idx] = bodies[idx].fx;
				//fy[idx] = bodies[j].fx;
				//f[idx].fx += F * dx / dist;
				//f[idx].fy += F * dy / dist;
				//bodies_return[idx].fx += F * dx / dist;
				//bodies_return[idx].fy += F * dy / dist;
			}
		}
		//bodies[idx].vx += dt * fx / bodies[idx].mass;
		//bodies[idx].vy += dt * fy / bodies[idx].mass;
		//bodies[idx].rx += dt * bodies[idx].vx;
		//bodies[idx].ry += dt * bodies[idx].vy;

		// Update velocity and integrate
		//returned_bodies[idx].vx += dt * returned_bodies[idx].fx / returned_bodies[idx].mass;
		//returned_bodies[idx].vy += dt * returned_bodies[idx].fy / returned_bodies[idx].mass;
		//returned_bodies[idx].rx += dt * returned_bodies[idx].vx;
		//returned_bodies[idx].ry += dt * returned_bodies[idx].vy;
		//vx[idx] += dt * fx / bodies[idx].mass;
		//vy[idx] += dt * fy / bodies[idx].mass;
	}

	//bodies_return[idx].vx += dt * bodies_return[idx].fx / bodies_return[idx].mass;
	//bodies_return[idx].vy += dt * bodies_return[idx].fy / bodies_return[idx].mass;
	//bodies_return[idx].rx += dt * bodies_return[idx].vx;
	//bodies_return[idx].ry += dt * bodies_return[idx].vy;
	//c->fx += F * dx / dist;
	//c->fy += F * dy / dist;
}

__global__ void add_force_simple_struct(Body_struct *bodies, double *r_rx, double *r_ry, const double G, int N, double dt)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < N)
	{
		double _fx, _fy = 0;
		for (int j = 0; j < N; j++)
		{
			if (idx != j)
			{
				double EPS = 3E4;      // softening parameter (just to avoid infinities)
				double dx = bodies[j].rx - bodies[idx].rx;
				double dy = bodies[j].ry - bodies[idx].ry;
				double dist = sqrt(dx*dx + dy*dy);
				double F = (G * bodies[idx].mass * bodies[j].mass) / (dist*dist + EPS*EPS);

				_fx += F * dx / dist;
				_fy += F * dy / dist;
			}
			__syncthreads();
		}
		bodies[idx].vx += dt * _fx / bodies[idx].mass;
		bodies[idx].vy += dt * _fy / bodies[idx].mass;
		r_rx[idx] += dt * bodies[idx].vx;
		r_ry[idx] += dt * bodies[idx].vy;
	}
}



__global__ void add_force_simple_doubles(double *vx, double *vy, const double *rx, const double *ry,
	const double *mass, double *r_rx, double *r_ry, const double G, int N, double dt)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < N)
	{
		double _fx, _fy = 0;
		//__syncthreads();
		#pragma unroll
		for (int j = 0; j < N; j++)
		{
			if (idx != j)
			{
				double EPS = 3E4;      // softening parameter (just to avoid infinities)
				double dx = rx[j] - rx[idx];
				double dy = ry[j] - ry[idx];
				double dist = sqrt(dx*dx + dy*dy);
				double F = (G * mass[idx] * mass[j]) / (dist*dist + EPS*EPS);

				_fx += F * dx / dist;
				_fy += F * dy / dist;
				//
			}
			//__syncthreads();
		}
		__syncthreads();
		vx[idx] += dt * _fx / mass[idx];
		vy[idx] += dt * _fy / mass[idx];
		r_rx[idx] += dt * vx[idx];
		r_ry[idx] += dt * vy[idx];
	}
}

double random()
{
	return static_cast <double> (rand()) / static_cast <double> (RAND_MAX);
}

int sgn(double d) {
	return d<-DBL_EPSILON ? -1 : d>DBL_EPSILON;
}

// The bodies are initialized in circular orbits around the central mass.
// This is just some physics to do that
double circlev(double rx, double ry)
{
	double solarmass = 1.98892e30;
	double r2 = sqrt(rx*rx + ry*ry);
	double numerator = (6.67e-11)*1e6*solarmass;
	return sqrt(numerator / r2);
}

// Initialize N bodies with random positions and circular velocities
void startthebodies(int N)
{
	double radius = 1e18;        // radius of universe
	double solarmass = 1.98892e30;
	for (int i = 0; i < N; i++) {
		double px = 1e18*exp(-1.8)*(.5 - random());
		double py = 1e18*exp(-1.8)*(.5 - random());
		double magv = circlev(px, py);

		double absangle = atan(abs(py / px));
		double thetav = M_PI / 2 - absangle;
		double phiv = random() * M_PI;
		double vx = -1 * sgn(py)*cos(thetav)*magv;
		double vy = sgn(px)*sin(thetav)*magv;
		// Orient a random 2D circular orbit
		if (random() <= .5) {
			vx = -vx;
			vy = -vy;
		}

		double mass = random() * solarmass * 10 + 1e20;
		// Color the masses in green gradients by mass
		int red = (int)floor(mass * 254 / (solarmass * 10 + 1e20));
		int blue = (int)floor(mass * 254 / (solarmass * 10 + 1e20));
		int green = 255;
		ALLEGRO_COLOR color = al_map_rgb(red, green, blue);
		// put a heavy body in the center
		if (i == 0)
			bodies.push_back(new Body(0, 0, 0, 0, 1e6*solarmass, color));

		bodies.push_back(new Body(px, py, vx, vy, mass, color));
	}
}

// Use the method in Body to reset the forces, then add all the new forces
void addforces(int N)
{
	for (int i = 0; i < N; i++) {
		bodies[i]->resetForce();
		// Notice-2 loops-->N^2 complexity
		for (int j = 0; j < N; j++) {
			if (i != j) bodies[i]->addForce(*bodies[j]);
		}
	}
	// Then, loop again and update the bodies using timestep dt
	for (int i = 0; i < N; i++) {
		bodies[i]->update(1e11);
	}
}

void draw_bodies()
{
	for (int i = 0; i<N; i++) {
		al_draw_circle((screen_size_x / 2) + (int)round(bodies[i]->rx / 1e18), (screen_size_y / 2) + (int)round(bodies[i]->ry / 1e18), 1.0f, bodies[i]->color, 0.75f);
		//al_draw_circle((screen_size_x / 2) + (int)round(rx[i] / 1e18), (screen_size_y / 2) + (int)round(ry[i] / 1e18), 1.0f, bodies[i]->color, 0.75f);
	}
}

int main(int argc, char **argv)
{

	ALLEGRO_DISPLAY *display = NULL;

	if (!al_init()) {
		fprintf(stderr, "failed to initialize allegro!\n");
		return -1;
	}

	display = al_create_display(screen_size_x, screen_size_y);
	if (!display) {
		fprintf(stderr, "failed to create display!\n");
		return -1;
	}

	al_clear_to_color(al_map_rgb(0, 0, 0));

	al_flip_display();
	al_init_primitives_addon();
	al_init_font_addon(); // initialize the font addon
	al_init_ttf_addon();// initialize the ttf (True Type Font) addon

	ALLEGRO_FONT *font = al_load_ttf_font("../Consolas.ttf", 24, 0);

	if (!font) {
		fprintf(stderr, "Could not load 'Consolas.ttf'.\n");
		return -1;
	}
	startthebodies(N);

	// Initialise CUDA - select device
	hipSetDevice(0);

	// Create host memory
	//auto body_data_size = sizeof(Body*) * N;
	//auto returned_body_data_Size = sizeof(Body) * N;
	//auto force_data_size = sizeof(force*) * N;

	auto double_data_size = sizeof(double) * N;
	//auto body_struct_data_size = sizeof(Body_struct) * N;

	//vector<Body> returned_bodies(N);
	
	//for (size_t i = 0; i < N; i++)
	//{
	//	returned_bodies.push_back(*bodies[i]);
	//}

	vector<double> vx(N);
	vector<double> vy(N);
	vector<double> rx(N);
	vector<double> ry(N);
	vector<double> fx(N);
	vector<double> fy(N);
	vector<double> mass(N);
	//
	for (size_t i = 0; i < N; i++)
	{
		// Init velocity
		vx[i] = bodies[i]->vx;
		vy[i] = bodies[i]->vy;
		// Init positions
		rx[i] = bodies[i]->rx;
		ry[i] = bodies[i]->ry;
		// Init forces
		//fx[i] = bodies[i]->fx;
		//fy[i] = bodies[i]->fy;
		// Init mass
		mass[i] = bodies[i]->mass;
	}
	//vector<Body_struct> bs(N);
	//for (size_t i = 0; i < N; i++)
	//{
	//	// Init velocity
	//	bs[i].vx = bodies[i]->vx;
	//	bs[i].vy = bodies[i]->vy;
	//	// Init positions
	//	bs[i].rx = bodies[i]->rx;
	//	bs[i].ry = bodies[i]->ry;
	//	// Init mass
	//	bs[i].mass = bodies[i]->mass;
	//}

	//vector<force*> f(N); // Output array
	vector<double> r_rx(N);
	vector<double> r_ry(N);
	//
	//for (size_t i = 0; i < N; i++)
	//{
	//	f[i] = (force*)malloc(force_data_size);
	//	f[i]->fx = 0.0;
	//	f[i]->fy = 0.0;
	//}
	
	// Declare buffers
	//Body *d_a;// = (Body*)malloc(body_data_size);
	////force *d_b;
	////double *d_b;
	//Body *d_b;
	//double *d_c;

	double *d_vx;
	double *d_vy;
	double *d_rx;
	double *d_ry;
	double *d_m;
	double *d_r_rx;
	double *d_r_ry;
	//Body_struct *d_bs;

	// Initialise buffers
	//hipMalloc((void**)&d_a, body_data_size);
	//hipMalloc((void**)&d_b, returned_body_data_Size);
	//hipMalloc((void**)&d_c, double_data_size);

	hipMalloc((void**)&d_vx, double_data_size);
	hipMalloc((void**)&d_vy, double_data_size);
	hipMalloc((void**)&d_rx, double_data_size);
	hipMalloc((void**)&d_ry, double_data_size);
	hipMalloc((void**)&d_m, double_data_size);
	hipMalloc((void**)&d_r_rx, double_data_size);
	hipMalloc((void**)&d_r_ry, double_data_size);

	//hipMalloc((void**)&d_bs, body_struct_data_size);


	//vx = (double*)malloc(double_data_size);
	//vy = (double*)malloc(double_data_size);

	//hipMalloc((void**)&buffer_C, double_data_size);

	int nBlocks = N / BLOCK_SIZE;
	//int nBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
	std::cout << "Number of blocks: " << nBlocks << ", block size: " << BLOCK_SIZE << std::endl;
	// Get the start time
	auto start = std::chrono::system_clock::now();
	double dt = 1e11;
	for (int sim_iterations = 0; sim_iterations <= 5000; sim_iterations++)
	{
		//addforces(N);
		//hipMemcpy(d_a, &bodies[0], body_data_size, hipMemcpyHostToDevice);

		hipMemcpy(d_vx, &vx[0], double_data_size, hipMemcpyHostToDevice);
		hipMemcpy(d_vy, &vy[0], double_data_size, hipMemcpyHostToDevice);
		hipMemcpy(d_rx, &rx[0], double_data_size, hipMemcpyHostToDevice);
		hipMemcpy(d_ry, &ry[0], double_data_size, hipMemcpyHostToDevice);
		hipMemcpy(d_m, &mass[0], double_data_size, hipMemcpyHostToDevice);
		//hipMemcpy(d_bs, &bs[0], body_struct_data_size, hipMemcpyHostToDevice);
		// Max 1024 threads per block
		
		add_force_simple_doubles<<<nBlocks, BLOCK_SIZE>>>(d_vx, d_vy, d_rx, d_ry, d_m, d_r_rx, d_r_ry, G, N, dt);
		//add_force_simple_struct << <nBlocks, BLOCK_SIZE >> >(d_bs, d_r_rx, d_r_ry, G, N, dt);
		
		//kernel << <1, 1 >> > ();
		// Wait for kernel to complete
		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if (error != hipSuccess)
		{
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
			exit(-1);
		}
		// Read output buffers back to the host
		//hipMemcpy(f[0], d_b, force_data_size, hipMemcpyDeviceToHost);
		//hipMemcpy(&returned_bodies[0], d_b, returned_body_data_Size, hipMemcpyDeviceToHost);
		//hipMemcpy(&vx[0], d_b, double_data_size, hipMemcpyDeviceToHost);
		//hipMemcpy(&vy[0], d_c, double_data_size, hipMemcpyDeviceToHost);
		hipMemcpy(&r_rx[0], d_r_rx, double_data_size, hipMemcpyDeviceToHost);
		hipMemcpy(&r_ry[0], d_r_ry, double_data_size, hipMemcpyDeviceToHost);

		//hipMemcpy(&fy[0], buffer_C, double_data_size, hipMemcpyDeviceToHost);
		error = hipGetLastError();
		if (error != hipSuccess)
		{
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
			exit(-1);
		}
		for (int i = 0; i < N; i++) 
		{
			//bodies[i]->fx = f[i]->fx;
			//bodies[i]->fy = f[i]->fy;
			//returned_bodies[i].update(1e11);
			//bodies[i]->update(1e11);
			bodies[i]->rx = r_rx[i];
			bodies[i]->ry = r_ry[i];
			// Integrate
			//bodies[i]->rx += dt * vx[i];
			//bodies[i]->ry += dt * vy[i];
		}

		draw_bodies();
		al_flip_display();
		al_clear_to_color(al_map_rgb(0, 0, 0));
		char buffer[30];
		_itoa_s(sim_iterations, buffer, 10);
		string s = "Simulation iterations: ";
		s += buffer;
		al_draw_text(font, al_map_rgb(255, 255, 255), 0, 0, ALLEGRO_ALIGN_LEFT, s.c_str());
	}

	// Get the end time
	auto end = std::chrono::system_clock::now();
	// Get the total time
	auto total = end - start;

	cout << "Time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(total).count() << " ms" << endl;

	al_destroy_display(display);

	// Clean up resources
	hipFree(d_vx);
	hipFree(d_vy);
	hipFree(d_rx);
	hipFree(d_ry);
	hipFree(d_m);
	//hipFree(d_bs);

	return 0;
}