#include "hip/hip_runtime.h"
// Translated into C++ from Java, based on the code available at: http://physics.princeton.edu/~fpretori/Nbody/
#define _USE_MATH_DEFINES
#include <cmath>
#include "hip/hip_runtime.h"
#include ""
#include "body.h"
#include <vector>
#include <allegro5/allegro.h>
#include <allegro5/allegro_primitives.h>
#include <allegro5/allegro_color.h>
#include <allegro5/allegro_font.h>
#include <allegro5/allegro_ttf.h>
#include <iostream>
#include <chrono>
#include <string>
using namespace std;
const double G = 6.673e-11;   // gravitational constant
int N = 1024;
vector<Body*> bodies; //Body bodies[1000];

int screen_size_x = 1024;
int screen_size_y = 768;
#define BLOCK_SIZE 256

struct force
{
	double fx;
	double fy;
};

__global__ void add_force(const Body *bodies, double *vx, double *vy, const double G, int N, double dt)
{
	// Get block index
	unsigned int block_idx = blockIdx.x;
	// Get thread index
	unsigned int thread_idx = threadIdx.x;
	// Get the number of threads per block
	unsigned int block_dim = blockDim.x;
	// Get the thread's unique ID = (block_idx * block_dim) + thread_idx;
	unsigned int idx = (block_idx * block_dim) + thread_idx;
	if (idx < N)
	{
		// Reset forces
		double fx, fy = 0.0;
		//bodies[idx].fx = 0.0;
		//bodies[idx].fy = 0.0;

		for (int tile = 0; tile < gridDim.x; tile++)
		{
			__shared__ double s_rx[BLOCK_SIZE];
			__shared__ double s_ry[BLOCK_SIZE];
			double t_rx = bodies[tile * blockDim.x + threadIdx.x].rx;
			double t_ry = bodies[tile * blockDim.x + threadIdx.x].ry;
			s_rx[threadIdx.x] = t_rx;
			s_ry[threadIdx.x] = t_ry;
			__syncthreads();
			for (int j = 0; j < BLOCK_SIZE; j++)
			{
				if (idx != j)
				{
					double EPS = 3E4;      // softening parameter (just to avoid infinities)
					double dx = s_rx[j] - bodies[idx].rx;
					double dy = s_ry[j] - bodies[idx].ry;
					double dist = sqrt(dx*dx + dy*dy);
					double F = (G * bodies[idx].mass * bodies[j].mass) / (dist*dist + EPS*EPS);

					// is this right? probably
					//f[idx].fx += F * dx / dist;
					//f[idx].fy += F * dy / dist;

					fx += F * dx / dist;
					fy += F * dy / dist;
				}
			}
			__syncthreads();
		}
		
		// Calculate velocity
		vx[idx] += dt * fx / bodies[idx].mass;
		vy[idx] += dt * fy / bodies[idx].mass;
	}
	//for (int j = 0; j < N; j++) 
	//{
	//	if (idx != j)
	//	{
	//		double EPS = 3E4;      // softening parameter (just to avoid infinities)
	//		double dx = bodies[j].rx - bodies[idx].rx;
	//		double dy = bodies[j].ry - bodies[idx].ry;
	//		double dist = sqrt(dx*dx + dy*dy);
	//		double F = (G * bodies[idx].mass * bodies[j].mass) / (dist*dist + EPS*EPS);

	//		//fx[idx] = bodies[idx].fx;
	//		//fy[idx] = bodies[j].fx;
	//		f[idx].fx += F * dx / dist;
	//		f[idx].fy += F * dy / dist;
	//		//bodies_return[idx].fx += F * dx / dist;
	//		//bodies_return[idx].fy += F * dy / dist;
	//	}
	//}

	// Update velocity
	//bodies_return[idx].vx += dt * bodies_return[idx].fx / bodies_return[idx].mass;
	//bodies_return[idx].vy += dt * bodies_return[idx].fy / bodies_return[idx].mass;
	//bodies_return[idx].rx += dt * bodies_return[idx].vx;
	//bodies_return[idx].ry += dt * bodies_return[idx].vy;
	//c->fx += F * dx / dist;
	//c->fy += F * dy / dist;
}

__global__ void add_force_simple(const Body *bodies, double *vx, double *vy, const double G, int N, double dt)
{
	// Get block index
	unsigned int block_idx = blockIdx.x;
	// Get thread index
	unsigned int thread_idx = threadIdx.x;
	// Get the number of threads per block
	unsigned int block_dim = blockDim.x;
	// Get the thread's unique ID = (block_idx * block_dim) + thread_idx;
	unsigned int idx = (block_idx * block_dim) + thread_idx;

	if (idx < N)
	{
		double fx, fy = 0;
		for (int j = 0; j < N; j++)
		{
			if (idx != j)
			{
				double EPS = 3E4;      // softening parameter (just to avoid infinities)
				double dx = bodies[j].rx - bodies[idx].rx;
				double dy = bodies[j].ry - bodies[idx].ry;
				double dist = sqrt(dx*dx + dy*dy);
				double F = (G * bodies[idx].mass * bodies[j].mass) / (dist*dist + EPS*EPS);

				fx += F * dx / dist;
				fy += F * dy / dist;
				//bodies[idx].fx = fx;
				//bodies[idx].fy = fy;
				//fx[idx] = bodies[idx].fx;
				//fy[idx] = bodies[j].fx;
				//f[idx].fx += F * dx / dist;
				//f[idx].fy += F * dy / dist;
				//bodies_return[idx].fx += F * dx / dist;
				//bodies_return[idx].fy += F * dy / dist;
			}
		}
		//bodies[idx].vx += dt * fx / bodies[idx].mass;
		//bodies[idx].vy += dt * fy / bodies[idx].mass;
		//bodies[idx].rx += dt * bodies[idx].vx;
		//bodies[idx].ry += dt * bodies[idx].vy;

		// Update velocity
		vx[idx] += dt * fx / bodies[idx].mass;
		vy[idx] += dt * fy / bodies[idx].mass;
	}

	//bodies_return[idx].vx += dt * bodies_return[idx].fx / bodies_return[idx].mass;
	//bodies_return[idx].vy += dt * bodies_return[idx].fy / bodies_return[idx].mass;
	//bodies_return[idx].rx += dt * bodies_return[idx].vx;
	//bodies_return[idx].ry += dt * bodies_return[idx].vy;
	//c->fx += F * dx / dist;
	//c->fy += F * dy / dist;
}



double random()
{
	return static_cast <double> (rand()) / static_cast <double> (RAND_MAX);
}

int sgn(double d) {
	return d<-DBL_EPSILON ? -1 : d>DBL_EPSILON;
}

// The bodies are initialized in circular orbits around the central mass.
// This is just some physics to do that
double circlev(double rx, double ry)
{
	double solarmass = 1.98892e30;
	double r2 = sqrt(rx*rx + ry*ry);
	double numerator = (6.67e-11)*1e6*solarmass;
	return sqrt(numerator / r2);
}

// Initialize N bodies with random positions and circular velocities
void startthebodies(int N)
{
	double radius = 1e18;        // radius of universe
	double solarmass = 1.98892e30;
	for (int i = 0; i < N; i++) {
		double px = 1e18*exp(-1.8)*(.5 - random());
		double py = 1e18*exp(-1.8)*(.5 - random());
		double magv = circlev(px, py);

		double absangle = atan(abs(py / px));
		double thetav = M_PI / 2 - absangle;
		double phiv = random() * M_PI;
		double vx = -1 * sgn(py)*cos(thetav)*magv;
		double vy = sgn(px)*sin(thetav)*magv;
		// Orient a random 2D circular orbit
		if (random() <= .5) {
			vx = -vx;
			vy = -vy;
		}

		double mass = random() * solarmass * 10 + 1e20;
		// Color the masses in green gradients by mass
		int red = (int)floor(mass * 254 / (solarmass * 10 + 1e20));
		int blue = (int)floor(mass * 254 / (solarmass * 10 + 1e20));
		int green = 255;
		ALLEGRO_COLOR color = al_map_rgb(red, green, blue);
		// put a heavy body in the center
		if (i == 0)
			bodies.push_back(new Body(0, 0, 0, 0, 1e6*solarmass, color));

		bodies.push_back(new Body(px, py, vx, vy, mass, color));
	}
}

// Use the method in Body to reset the forces, then add all the new forces
void addforces(int N)
{
	for (int i = 0; i < N; i++) {
		bodies[i]->resetForce();
		// Notice-2 loops-->N^2 complexity
		for (int j = 0; j < N; j++) {
			if (i != j) bodies[i]->addForce(*bodies[j]);
		}
	}
	// Then, loop again and update the bodies using timestep dt
	for (int i = 0; i < N; i++) {
		bodies[i]->update(1e11);
	}
}

void draw_bodies()
{
	for (int i = 0; i<N; i++) {
		al_draw_circle((screen_size_x / 2) + (int)round(bodies[i]->rx / 1e18), (screen_size_y / 2) + (int)round(bodies[i]->ry / 1e18), 1.0f, bodies[i]->color, 0.75f);
	}
}

int main(int argc, char **argv)
{

	ALLEGRO_DISPLAY *display = NULL;

	if (!al_init()) {
		fprintf(stderr, "failed to initialize allegro!\n");
		return -1;
	}

	display = al_create_display(screen_size_x, screen_size_y);
	if (!display) {
		fprintf(stderr, "failed to create display!\n");
		return -1;
	}

	al_clear_to_color(al_map_rgb(0, 0, 0));

	al_flip_display();
	al_init_primitives_addon();
	al_init_font_addon(); // initialize the font addon
	al_init_ttf_addon();// initialize the ttf (True Type Font) addon

	ALLEGRO_FONT *font = al_load_ttf_font("../Consolas.ttf", 24, 0);

	if (!font) {
		fprintf(stderr, "Could not load 'Consolas.ttf'.\n");
		return -1;
	}
	startthebodies(N);

	// Initialise CUDA - select device
	hipSetDevice(0);

	// Create host memory
	auto body_data_size = sizeof(Body*) * N;
	//auto force_data_size = sizeof(force*) * N;
	auto double_data_size = sizeof(double) * N;
	vector<double> vx(N);
	vector<double> vy(N);
	//vector<force*> f(N); // Output array
	//
	//for (size_t i = 0; i < N; i++)
	//{
	//	f[i] = (force*)malloc(force_data_size);
	//	f[i]->fx = 0.0;
	//	f[i]->fy = 0.0;
	//}
	
	// Declare buffers
	Body *d_a;
	//force *d_b;
	double *d_b;
	double *d_c;

	// Initialise buffers
	hipMalloc((void**)&d_a, body_data_size);
	hipMalloc((void**)&d_b, double_data_size);
	hipMalloc((void**)&d_c, double_data_size);

	//vx = (double*)malloc(double_data_size);
	//vy = (double*)malloc(double_data_size);

	//hipMalloc((void**)&buffer_C, double_data_size);

	//int nBlocks = N / BLOCK_SIZE;
	int nBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

	// Get the start time
	auto start = std::chrono::system_clock::now();
	double dt = 1e11;
	for (int sim_iterations = 0; sim_iterations <= 5000; sim_iterations++)
	{
		//addforces(N);
		hipMemcpy(d_a, &bodies[0], body_data_size, hipMemcpyHostToDevice);

		// Max 1024 threads per block
		add_force<<<nBlocks, BLOCK_SIZE>>>(d_a, d_b, d_c, G, N, dt);
		//kernel << <1, 1 >> > ();
		// Wait for kernel to complete
		hipDeviceSynchronize();

		// Read output buffers back to the host
		//hipMemcpy(f[0], d_b, force_data_size, hipMemcpyDeviceToHost);
		hipMemcpy(&vx[0], d_b, double_data_size, hipMemcpyDeviceToHost);
		hipMemcpy(&vy[0], d_c, double_data_size, hipMemcpyDeviceToHost);
		//hipMemcpy(&fy[0], buffer_C, double_data_size, hipMemcpyDeviceToHost);

		for (int i = 0; i < N; i++) 
		{
			//bodies[i]->fx = f[i]->fx;
			//bodies[i]->fy = f[i]->fy;

			//bodies[i]->update(1e11);

			// Integrate
			bodies[i]->rx += dt * vx[i];
			bodies[i]->ry += dt * vy[i];
		}

		draw_bodies();
		al_flip_display();
		al_clear_to_color(al_map_rgb(0, 0, 0));
		char buffer[30];
		_itoa_s(sim_iterations, buffer, 10);
		string s = "Simulation iterations: ";
		s += buffer;
		al_draw_text(font, al_map_rgb(255, 255, 255), 0, 0, ALLEGRO_ALIGN_LEFT, s.c_str());
	}

	// Get the end time
	auto end = std::chrono::system_clock::now();
	// Get the total time
	auto total = end - start;

	cout << "Time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(total).count() << " ms" << endl;

	al_destroy_display(display);

	// Clean up resources
	hipFree(d_a);
	hipFree(d_b);
	//hipFree(buffer_C);

	return 0;
}